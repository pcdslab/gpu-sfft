/*
 *  Copyright (C) 2019 Oswaldo Artiles and Fahad Saeed 
 *  Florida International University, Florida, USA.
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * as published by the Free Software Foundation; either version 3
 * of the License, or (at your option) any later version.
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE   
 * Please refer to the GNU General Public License for more details.
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
 */

//includes CUDA project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

extern "C"{
         #include "cudaFft.h"
         #include "utils.h"
}
/* Compute and returns the FFT of the input signal
 * using cuFFT(NVIDIA-CUDA) library.
 * Prints the timing of the FFT transformation. 
 */ 
int cuda_fft_dft_d(complex_t *h_input,  complex_t *h_output, int nx, int forward, int print_y){

  int batch = 1;

  /*timing variables */
  float time_create_plan;
  float time_transf_H_to_D;
  float time_transf_D_to_H;
  float time_execution;
  float time_total;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  /*Allocate device  memory  for the input signal*/
  complex_t *d_input;
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_input), sizeof(hipfftDoubleComplex)*nx*batch));

  /* Copy host memory to device memory*/
  hipEventRecord(start);
  checkCudaErrors(hipMemcpy(d_input, h_input, nx*batch*sizeof(complex_t),hipMemcpyHostToDevice));
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time_transf_H_to_D,start, stop);

  /*cuFFT plan creation*/
  hipfftHandle plan;
  hipEventRecord(start);
  checkCudaErrors(hipfftPlan1d(&plan,nx,HIPFFT_Z2Z,batch));
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time_create_plan,start, stop);

  /*cuFFT plan  execution*/
  hipEventRecord(start);
  checkCudaErrors(hipfftExecZ2Z(plan,reinterpret_cast<hipfftDoubleComplex *>(d_input),
			       reinterpret_cast<hipfftDoubleComplex *>(d_input),forward ? HIPFFT_FORWARD:HIPFFT_BACKWARD));
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time_execution,start, stop);

  /*Copy device memory to host memory*/
  hipEventRecord(start);
  checkCudaErrors(hipMemcpy(h_output, d_input, nx*batch*sizeof(complex_t),hipMemcpyDeviceToHost));
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time_transf_D_to_H,start, stop);

  /* Destroy cuFFT context*/
  checkCudaErrors(hipfftDestroy(plan));

  /*cleanup memory*/
  checkCudaErrors(hipFree(d_input));
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));

  time_total = time_transf_H_to_D + time_execution + time_transf_D_to_H;
  print_timing (time_create_plan,  time_transf_H_to_D, time_transf_D_to_H,
		time_execution, time_total, print_y);


  return 0;

}// end cuda_fft_dft_d


void print_timing (float time_create_plan, float time_transf_H_to_D,
		   float time_transf_D_to_H, float time_execution,
		   float time_total, int print_y){

  if (print_y) {
    printf("Memory transfer host to device execution time %4.2fmilisec\n",time_transf_H_to_D);
    printf("Plan creation execution time %4.2fms\n", time_create_plan);
    printf("cuFFT execution time    %4.2fms\n", time_execution);
    printf("Memory transfer device to host execution time %4.2fms\n",time_transf_D_to_H);
    printf("total time %4.2fms\n",time_total);
  }

}//end print_timing
